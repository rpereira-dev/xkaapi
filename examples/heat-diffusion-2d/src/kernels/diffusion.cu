#include "hip/hip_runtime.h"
/*
** Copyright 2024 INRIA
**
** Contributors :
**
** Romain PEREIRA, romain.pereira@inria.fr
** Romain PEREIRA, rpereira@anl.gov
** This software is a computer program whose purpose is to execute
** blas subroutines on multi-GPUs system.
**
** This software is governed by the CeCILL-C license under French law and
** abiding by the rules of distribution of free software.  You can  use,
** modify and/ or redistribute the software under the terms of the CeCILL-C
** license as circulated by CEA, CNRS and INRIA at the following URL
** "http://www.cecill.info".

** As a counterpart to the access to the source code and  rights to copy,
** modify and redistribute granted by the license, users are provided only
** with a limited warranty  and the software's author,  the holder of the
** economic rights,  and the successive licensors  have only  limited
** liability.

** In this respect, the user's attention is drawn to the risks associated
** with loading,  using,  modifying and/or developing or reproducing the
** software by the user in light of its specific status of free software,
** that may mean  that it is complicated to manipulate,  and  that  also
** therefore means  that it is reserved for developers  and  experienced
** professionals having in-depth computer knowledge. Users are therefore
** encouraged to load and test the software's suitability as regards their
** requirements in conditions enabling the security of their systems and/or
** data to be ensured and,  more generally, to use and operate it in the
** same conditions as regards security.

** The fact that you are presently reading this means that you have had
** knowledge of the CeCILL-C license and that you accept its terms.
**/

// from GPT4.o

# if 0
// Optimizing a CUDA stencil kernel involves several strategies, including improving memory access patterns, using shared memory, and optimizing thread and block configurations. Here's a revised version of your kernel with some optimizations:

__global__ void diffusion_cuda_kernel(TYPE *src, int ld_src, TYPE *dst, int ld_dst, int tile_x, int tile_y, int tsx, int tsy) {
    // Calculate global indices
    const int li = threadIdx.x;
    const int lj = threadIdx.y;
    const int i = tile_x * tsx + blockIdx.x * blockDim.x + li;
    const int j = tile_y * tsy + blockIdx.y * blockDim.y + lj;

    // Shared memory for the current tile
    extern __shared__ TYPE shared_mem[];

    // Calculate shared memory indices
    const int shared_i = li + 1;
    const int shared_j = lj + 1;

    // Load data into shared memory
    if (i < NX && j < NY) {
        shared_mem[shared_j * (blockDim.x + 2) + shared_i] = GRID(src, i, j, ld_src);

        // Load halo elements
        if (li == 0 && i > 0) {
            shared_mem[shared_j * (blockDim.x + 2)] = GRID(src, i - 1, j, ld_src);
        }
        if (li == blockDim.x - 1 && i < NX - 1) {
            shared_mem[shared_j * (blockDim.x + 2) + shared_i + 1] = GRID(src, i + 1, j, ld_src);
        }
        if (lj == 0 && j > 0) {
            shared_mem[shared_i] = GRID(src, i, j - 1, ld_src);
        }
        if (lj == blockDim.y - 1 && j < NY - 1) {
            shared_mem[(shared_j + 1) * (blockDim.x + 2) + shared_i] = GRID(src, i, j + 1, ld_src);
        }
    }

    __syncthreads();

    // Perform computation if within bounds
    if (i > 0 && i < NX - 1 && j > 0 && j < NY - 1) {
        GRID(dst, i, j, ld_dst) = shared_mem[shared_j * (blockDim.x + 2) + shared_i] + ALPHA * DT / (DX * DY) * (
            (shared_mem[shared_j * (blockDim.x + 2) + shared_i + 1] - 2 * shared_mem[shared_j * (blockDim.x + 2) + shared_i] + shared_mem[shared_j * (blockDim.x + 2) + shared_i - 1]) / (DX * DX) +
            (shared_mem[(shared_j + 1) * (blockDim.x + 2) + shared_i] - 2 * shared_mem[shared_j * (blockDim.x + 2) + shared_i] + shared_mem[(shared_j - 1) * (blockDim.x + 2) + shared_i]) / (DY * DY)
        );
    }
}
Key Optimizations:
Shared Memory Usage: The kernel now uses shared memory to store the tile being processed. This reduces global memory accesses, which are slower compared to shared memory accesses.

Halo Region: The kernel loads a halo region into shared memory to handle boundary conditions within the tile. This ensures that all necessary data for computation is available in shared memory.

Thread Synchronization: A __syncthreads() call is used to ensure all threads have loaded their data into shared memory before any computation begins.

Boundary Checks: The kernel checks if the current thread is within the valid computational domain before performing the stencil operation.

Thread and Block Configuration: Ensure that the block size is chosen to maximize occupancy while considering shared memory usage. A typical block size might be 16x16 or 32x32, but this should be tuned based on your specific hardware and problem size.

Additional Considerations:
Memory Coalescing: Ensure that global memory accesses are coalesced. This is generally achieved by having threads access contiguous memory locations.

Occupancy: Use CUDA occupancy calculator tools to determine the optimal block size and shared memory usage for your specific GPU.

# endif

# include <heat/consts.h>

/* A naive kernel to update the grid */
__global__
void
diffusion_cuda_kernel(TYPE * src, int ld_src, TYPE * dst, int ld_dst, int tile_x, int tile_y, int tsx, int tsy)
{
    const int li = blockIdx.x * blockDim.x + threadIdx.x;
    const int lj = blockIdx.y * blockDim.y + threadIdx.y;
    const int  i = tile_x * tsx + li;
    const int  j = tile_y * tsy + lj;

    // boundary conditions fixed
    if (i > 0 && i < NX - 1 && j > 0 && j < NY - 1)
    {
        GRID(dst, li, lj, ld_dst) = GRID(src, li, lj, ld_src) + ALPHA * DT / (DX * DY) * (
                (GRID(src, li+1,   lj, ld_src) - 2 * GRID(src, li, lj, ld_src) + GRID(src, li-1,   lj, ld_src)) / (DX * DX) +
                (GRID(src,   li, lj+1, ld_src) - 2 * GRID(src, li, lj, ld_src) + GRID(src,   li, lj-1, ld_src)) / (DY * DY)
            );
    }
}

extern "C"
void
diffusion_cuda(
    hipStream_t stream,
    TYPE * src, int ld_src,
    TYPE * dst, int ld_dst,
    int tile_x, int tile_y,
    int tsx, int tsy
) {
    // Number of threads per block line
    const unsigned int dtsx = (32 < (tsx) ? 32 : (tsx));
    const unsigned int dtsy = (32 < (tsy) ? 32 : (tsy));

    // how many threads we need in total
    dim3 T = {(unsigned int) tsx, (unsigned int) tsy, 1};

    // block dim
    dim3 B(dtsx, dtsy, 1);

    // grid
    dim3 G((T.x + B.x - 1) / B.x,  (T.y + B.y - 1) / B.y, 1);

    // kernel launch
    diffusion_cuda_kernel<<<G, B, 0, stream>>>(src, ld_src, dst, ld_dst, tile_x, tile_y, tsx, tsy);
}
